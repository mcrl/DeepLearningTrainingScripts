#include "hip/hip_runtime.h"
#include "embedding.h"

__global__ 
void forward_kernel (int *in, float *table, float *out, int batch_size, int vector_size) {
    int batch = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ( batch >= batch_size || y >= vector_size ) return;

    out[batch * vector_size + y] = table[in[batch] * vector_size + y];
}

__global__ 
void backward_kernel (int *in, float *table, float *out_grad, int batch_size, int vector_size, float lr) {
    int batch = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ( batch >= batch_size || y >= vector_size ) return;

    atomicAdd(table + in[batch] * vector_size + y, lr * out_grad[batch * vector_size + y]);
}


Embedding::Embedding (int batch_size_, int rows_, int vector_size_, int ndev_, bool init) :
                batch_size(batch_size_), rows(rows_), vector_size(vector_size_), ndev(ndev_)
{
    CUDA_CALL( hipMalloc(&table, rows * vector_size * sizeof(float)) );
    CUDA_CALL( hipMalloc(&gatheredIn, batch_size * NDEV * NNODE * sizeof(int)) );
    CUDA_CALL( hipMalloc(&gatheredDelta, batch_size * NDEV * NNODE * vector_size * sizeof(float)) );
    if ( init ) initRandUniform(table, rows * vector_size, sqrt(1.0 / rows), ndev);
}

void Embedding::forward (IntegerTensor *t_in, Tensor *t_out) {

    CUDA_CALL( hipSetDevice(ndev) );

    dim3 blocks((batch_size+15) / 16, (vector_size+15) / 16);
    dim3 threadPerBlock(16, 16);
    forward_kernel<<<blocks, threadPerBlock>>>(t_in->d_mem, table, t_out->d_mem, batch_size, vector_size);
}

void Embedding::backward (IntegerTensor *t_in, Tensor *t_out, Tensor *t_out_grad) {
    in = t_in->d_mem;
    delta = t_out_grad->d_mem;
}

void Embedding::update () {
    CUDA_CALL( hipSetDevice(ndev) );

    dim3 blocks((batch_size * NDEV * NNODE + 15) / 16, (vector_size+15) / 16);
    dim3 threadPerBlock(16, 16);
    backward_kernel<<<blocks, threadPerBlock>>>(gatheredIn, table, gatheredDelta, batch_size * NDEV, vector_size, lr);
}


